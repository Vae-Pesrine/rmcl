#include "hip/hip_runtime.h"
#include "rmcl/math/math.cuh"

#include <rmagine/math/math.cuh>

#include <rmagine/math/SVD.hpp>
#include <rmagine/util/prints.h>

using namespace rmagine;
namespace rm = rmagine;


namespace rmcl 
{

CorrectionCuda::CorrectionCuda(SVDCudaPtr svd)
:m_svd(svd) 
{

}

CorrectionCuda::CorrectionCuda()
:CorrectionCuda(std::make_shared<SVDCuda>())
{

}

CorrectionCuda::CorrectionCuda(
    CudaContextPtr ctx)
:CorrectionCuda(std::make_shared<SVDCuda>(ctx))
{

}

CorrectionCuda::CorrectionCuda(
    CudaStreamPtr stream)
:CorrectionCuda(std::make_shared<SVDCuda>(stream))
{

}

void CorrectionCuda::correction_from_covs(
    const MemoryView<Vector, VRAM_CUDA>& ms,
    const MemoryView<Vector, VRAM_CUDA>& ds,
    const MemoryView<Matrix3x3, VRAM_CUDA>& Cs,
    const MemoryView<unsigned int, VRAM_CUDA>& Ncorr,
    MemoryView<Transform, VRAM_CUDA>& Tdelta) const
{
    rm::Memory<rm::Matrix3x3, rm::VRAM_CUDA> Us(Cs.size());
    rm::Memory<rm::Matrix3x3, rm::VRAM_CUDA> Vs(Cs.size());
    // dont need this
    rm::Memory<rm::Vector, rm::VRAM_CUDA> Ss(Cs.size());

    m_svd->calcUSV(Cs, Us, Ss, Vs);
    
    // debug
    // {
    //     // Memory<Matrix3x3, RAM> Cs_ = Cs;
    //     // Memory<Matrix3x3, RAM> Us_(Cs.size());
    //     // Memory<Matrix3x3, RAM> Vs_(Cs.size());

    //     // static SVD svd;
    //     // svd.calcUV(Cs_, Us_, Vs_);

    //     // Us = Us_;
    //     // Vs = Vs_;
    //     Memory<Matrix3x3, RAM> Cs_       = Cs;
    //     Memory<Matrix3x3, RAM> Us_       = Us;
    //     Memory<Vector, RAM> ss_          = ss;
    //     Memory<Matrix3x3, RAM> Vs_       = Vs;
    //     Memory<unsigned int, RAM> Ncorr_ = Ncorr;

    //     std::cout << "C:" << std::endl;
    //     std::cout << Cs_[0] << std::endl;

    //     std::cout << "U:" << std::endl;
    //     std::cout << Us_[0] << std::endl;
    //     std::cout << "s:" << std::endl;
    //     std::cout << ss_[0] << std::endl;
    //     std::cout << "V:" << std::endl;
    //     std::cout << Vs_[0] << std::endl;
    //     std::cout << "Ncorr: " << std::endl;
    //     std::cout << Ncorr_[0] << std::endl;
        

    // }

    compute_transform(Us, Vs, ds, ms, Tdelta);
}

void CorrectionCuda::correction_from_covs(
    const MemoryView<Vector, VRAM_CUDA>& ms,
    const MemoryView<Vector, VRAM_CUDA>& ds,
    const MemoryView<Matrix3x3, VRAM_CUDA>& Cs,
    const MemoryView<unsigned int, VRAM_CUDA>& Ncorr,
    MemoryView<Quaternion, VRAM_CUDA>& Rdelta,
    MemoryView<Vector, VRAM_CUDA>& tdelta) const
{
    rm::Memory<rm::Matrix3x3, rm::VRAM_CUDA> Us(Cs.size());
    rm::Memory<rm::Matrix3x3, rm::VRAM_CUDA> Vs(Cs.size());

    m_svd->calcUV(Cs, Us, Vs);
    rm::transposeInplace(Vs);

    rm::multNxN(Us, Vs, Rdelta);
    rm::subNxN(ds, rm::multNxN(Rdelta, ms), tdelta);
}

void CorrectionCuda::correction_from_covs(
    const CorrectionPreResults<VRAM_CUDA>& pre_res,
    MemoryView<Transform, VRAM_CUDA>& Tdelta) const
{
    correction_from_covs(pre_res.ms, pre_res.ds, pre_res.Cs, pre_res.Ncorr, Tdelta);
}

Memory<Transform, VRAM_CUDA> CorrectionCuda::correction_from_covs(
    const CorrectionPreResults<VRAM_CUDA>& pre_res) const
{
    Memory<Transform, VRAM_CUDA> Tdelta(pre_res.ms.size());
    correction_from_covs(pre_res, Tdelta);
    return Tdelta;
}



__global__
void compute_transform_kernel(
    const rm::Matrix3x3* Us,
    const rm::Matrix3x3* Vs,
    const rm::Vector* ds,
    const rm::Vector* ms,
    rm::Transform* dT,
    unsigned int N)
{
    const unsigned int pid = blockIdx.x * blockDim.x + threadIdx.x;
    if(pid < N)
    {
        // input - read
        const rm::Matrix3x3 U = Us[pid];
        const rm::Matrix3x3 V = Vs[pid];
        const rm::Vector d = ds[pid];
        const rm::Vector m = ms[pid];

        // output
        rm::Transform T;
        rm::Matrix3x3 S = rm::Matrix3x3::Identity();
        if(U.det() * V.det() < 0)
        {
            S(2, 2) = -1;
        }

        // computation
        T.R.set(U * S * V.transpose());
        T.R.normalizeInplace();
        T.t = d - T.R * m;

        // write
        dT[pid] = T;
    }
}


void compute_transform(
    const rm::MemoryView<rm::Matrix3x3, rm::VRAM_CUDA>& Us,
    const rm::MemoryView<rm::Matrix3x3, rm::VRAM_CUDA>& Vs,
    const rm::MemoryView<rm::Vector, rm::VRAM_CUDA>& ds,
    const rm::MemoryView<rm::Vector, rm::VRAM_CUDA>& ms,
    rm::MemoryView<rmagine::Transform, rm::VRAM_CUDA>& dT)
{
    constexpr unsigned int blockSize = 1024;
    const unsigned int gridSize = (dT.size() + blockSize - 1) / blockSize;
    compute_transform_kernel<<<gridSize, blockSize>>>(Us.raw(), Vs.raw(), ds.raw(), ms.raw(), dT.raw(), dT.size());
}

// weighted average by
// - number of correspondences
// - fixed weights
// TODO: more than two


// template<typename T>
// __global__ void weighted_average_kernel(
//     const T* a, const unsigned int* Na,
//     const T* b, const unsigned int* Nb,
//     const unsigned int N, // elements
//     T* c, unsigned int* Nc)
// {
//     const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
//     if(id < N)
//     {
//         const unsigned int Ncorr_ = Na[id] + Nb[id];
//         const float Ncorrf = static_cast<float>(Ncorr_);
//         const float wa = static_cast<float>(Na[id]) / Ncorrf;
//         const float wb = static_cast<float>(Nb[id]) / Ncorrf;

//         c[id] = a[id] * wa + b[id] * wb;
//         Nc[id] = Ncorr_;
//     }
// }

__global__ void weighted_average_kernel(
    const Vector* ms1, const Vector* ds1, const Matrix3x3* Cs1, const unsigned int* Ncorr1,
    const Vector* ms2, const Vector* ds2, const Matrix3x3* Cs2, const unsigned int* Ncorr2,
    const unsigned int N, // Nelements
    Vector* ms, Vector* ds, Matrix3x3* Cs, unsigned int* Ncorr)
{
    const unsigned int pid = blockIdx.x * blockDim.x + threadIdx.x;
    if(pid < N)
    {
        const unsigned int Ncorr_ = Ncorr1[pid] + Ncorr2[pid];
        const float Ncorrf = static_cast<float>(Ncorr_);
        float w1 = static_cast<float>(Ncorr1[pid]) / Ncorrf;
        float w2 = static_cast<float>(Ncorr2[pid]) / Ncorrf;

        ms[pid] = ms1[pid] * w1 + ms2[pid] * w2;
        ds[pid] = ds1[pid] * w1 + ds2[pid] * w2;
        Cs[pid] = Cs1[pid] * w1 + Cs2[pid] * w2;
        Ncorr[pid] = Ncorr_;
    }
}


void weighted_average(
    const MemoryView<Vector, VRAM_CUDA>& ms1,
    const MemoryView<Vector, VRAM_CUDA>& ds1,
    const MemoryView<Matrix3x3, VRAM_CUDA>& Cs1,
    const MemoryView<unsigned int, VRAM_CUDA>& Ncorr1,
    const MemoryView<Vector, VRAM_CUDA>& ms2,
    const MemoryView<Vector, VRAM_CUDA>& ds2,
    const MemoryView<Matrix3x3, VRAM_CUDA>& Cs2,
    const MemoryView<unsigned int, VRAM_CUDA>& Ncorr2,
    MemoryView<Vector, VRAM_CUDA>& ms,
    MemoryView<Vector, VRAM_CUDA>& ds,
    MemoryView<Matrix3x3, VRAM_CUDA>& Cs,
    MemoryView<unsigned int, VRAM_CUDA>& Ncorr)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (ms1.size() + blockSize - 1) / blockSize;

    weighted_average_kernel<<<gridSize, blockSize>>>(
        ms1.raw(), ds1.raw(), Cs1.raw(), Ncorr1.raw(),
        ms2.raw(), ds2.raw(), Cs2.raw(), Ncorr2.raw(),
        ms1.size(),
        ms.raw(), ds.raw(), Cs.raw(), Ncorr.raw()
    );
}

__global__ void weighted_average_kernel(
    const Vector* ms1, const Vector* ds1, const Matrix3x3* Cs1, const unsigned int* Ncorr1, const float w1,
    const Vector* ms2, const Vector* ds2, const Matrix3x3* Cs2, const unsigned int* Ncorr2, const float w2,
    const unsigned int N, // Nelements
    Vector* ms, Vector* ds, Matrix3x3* Cs, unsigned int* Ncorr)
{
    const unsigned int pid = blockIdx.x * blockDim.x + threadIdx.x;
    if(pid < N)
    {
        const unsigned int Ncorr_ = Ncorr1[pid] + Ncorr2[pid];

        ms[pid] = ms1[pid] * w1 + ms2[pid] * w2;
        ds[pid] = ds1[pid] * w1 + ds2[pid] * w2;
        Cs[pid] = Cs1[pid] * w1 + Cs2[pid] * w2;
        Ncorr[pid] = Ncorr_;
    }
}

void weighted_average(
    const MemoryView<Vector, VRAM_CUDA>& ms1,
    const MemoryView<Vector, VRAM_CUDA>& ds1,
    const MemoryView<Matrix3x3, VRAM_CUDA>& Cs1,
    const MemoryView<unsigned int, VRAM_CUDA>& Ncorr1,
    float w1,
    const MemoryView<Vector, VRAM_CUDA>& ms2,
    const MemoryView<Vector, VRAM_CUDA>& ds2,
    const MemoryView<Matrix3x3, VRAM_CUDA>& Cs2,
    const MemoryView<unsigned int, VRAM_CUDA>& Ncorr2,
    float w2,
    MemoryView<Vector, VRAM_CUDA>& ms,
    MemoryView<Vector, VRAM_CUDA>& ds,
    MemoryView<Matrix3x3, VRAM_CUDA>& Cs,
    MemoryView<unsigned int, VRAM_CUDA>& Ncorr)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (ms1.size() + blockSize - 1) / blockSize;

    weighted_average_kernel<<<gridSize, blockSize>>>(
        ms1.raw(), ds1.raw(), Cs1.raw(), Ncorr1.raw(), w1,
        ms2.raw(), ds2.raw(), Cs2.raw(), Ncorr2.raw(), w2,
        ms1.size(),
        ms.raw(), ds.raw(), Cs.raw(), Ncorr.raw()
    );
}

void weighted_average(
    const std::vector<MemoryView<Vector, VRAM_CUDA> >& model_means,
    const std::vector<MemoryView<Vector, VRAM_CUDA> >& dataset_means,
    const std::vector<MemoryView<Matrix3x3, VRAM_CUDA> >& covs,
    const std::vector<MemoryView<unsigned int, VRAM_CUDA> >& Ncorrs,
    MemoryView<Vector, VRAM_CUDA>& ms,
    MemoryView<Vector, VRAM_CUDA>& ds,
    MemoryView<Matrix3x3, VRAM_CUDA>& Cs,
    MemoryView<unsigned int, VRAM_CUDA>& Ncorr)
{
    copy(model_means[0], ms);
    copy(dataset_means[0], ds);
    copy(covs[0], Cs);
    copy(Ncorrs[0], Ncorr);

    for(size_t i=1; i<model_means.size(); i++)
    {
        weighted_average(
            model_means[i], dataset_means[i], covs[i], Ncorrs[i],
            ms, ds, Cs, Ncorr,
            ms, ds, Cs, Ncorr);
    }
}

void weighted_average(
    const std::vector<MemoryView<Vector, VRAM_CUDA> >& model_means,
    const std::vector<MemoryView<Vector, VRAM_CUDA> >& dataset_means,
    const std::vector<MemoryView<Matrix3x3, VRAM_CUDA> >& covs,
    const std::vector<MemoryView<unsigned int, VRAM_CUDA> >& Ncorrs,
    const std::vector<float>& weights,
    MemoryView<Vector, VRAM_CUDA>& ms,
    MemoryView<Vector, VRAM_CUDA>& ds,
    MemoryView<Matrix3x3, VRAM_CUDA>& Cs,
    MemoryView<unsigned int, VRAM_CUDA>& Ncorr)
{
    ms = model_means[0];
    ds = dataset_means[0];
    Cs = covs[0];
    Ncorr = Ncorrs[0];

    float w = weights[0];

    for(size_t i=1; i<model_means.size(); i++)
    {
        weighted_average(
            model_means[i], dataset_means[i], covs[i], Ncorrs[i], weights[i],
            ms, ds, Cs, Ncorr, w,
            ms, ds, Cs, Ncorr);
        w = 1.0;
    }
}

void weighted_average(
    const std::vector<CorrectionPreResults<VRAM_CUDA> >& pre_results,
    CorrectionPreResults<VRAM_CUDA>& pre_results_combined)
{
    // std::cout << "wa2" << std::endl;
    // source: to fuse
    std::vector<MemoryView<Vector, VRAM_CUDA> > ms;
    std::vector<MemoryView<Vector, VRAM_CUDA> > ds;
    std::vector<MemoryView<Matrix3x3, VRAM_CUDA> > Cs;
    std::vector<MemoryView<unsigned int, VRAM_CUDA> > Ncorrs;

    for(size_t i = 0; i < pre_results.size(); i++)
    {
        ms.push_back(pre_results[i].ms);
        ds.push_back(pre_results[i].ds);
        Cs.push_back(pre_results[i].Cs);
        Ncorrs.push_back(pre_results[i].Ncorr);
    }

    weighted_average(ms, ds, Cs, Ncorrs, 
        pre_results_combined.ms, pre_results_combined.ds, pre_results_combined.Cs, pre_results_combined.Ncorr);
}

CorrectionPreResults<VRAM_CUDA> weighted_average(
    const std::vector<CorrectionPreResults<VRAM_CUDA> >& pre_results)
{
    CorrectionPreResults<rmagine::VRAM_CUDA> res;
    size_t Nposes = pre_results[0].Cs.size();

    res.ms.resize(Nposes);
    res.ds.resize(Nposes);
    res.Cs.resize(Nposes);
    res.Ncorr.resize(Nposes);

    weighted_average(pre_results, res);

    return res;
}

void weighted_average(
    const std::vector<CorrectionPreResults<VRAM_CUDA> >& pre_results,
    const std::vector<float>& weights,
    CorrectionPreResults<VRAM_CUDA>& pre_results_combined)
{
    // source: to fuse
    std::vector<MemoryView<Vector, VRAM_CUDA> > ms;
    std::vector<MemoryView<Vector, VRAM_CUDA> > ds;
    std::vector<MemoryView<Matrix3x3, VRAM_CUDA> > Cs;
    std::vector<MemoryView<unsigned int, VRAM_CUDA> > Ncorrs;

    for(size_t i = 0; i < pre_results.size(); i++)
    {
        ms.push_back(pre_results[i].ms);
        ds.push_back(pre_results[i].ds);
        Cs.push_back(pre_results[i].Cs);
        Ncorrs.push_back(pre_results[i].Ncorr);
    }

    weighted_average(ms, ds, Cs, Ncorrs, weights,
        pre_results_combined.ms, pre_results_combined.ds, pre_results_combined.Cs, pre_results_combined.Ncorr);
}

CorrectionPreResults<VRAM_CUDA> weighted_average(
    const std::vector<CorrectionPreResults<VRAM_CUDA> >& pre_results,
    const std::vector<float>& weights)
{
    CorrectionPreResults<rmagine::VRAM_CUDA> res;
    size_t Nposes = pre_results[0].Cs.size();

    res.ms.resize(Nposes);
    res.ds.resize(Nposes);
    res.Cs.resize(Nposes);
    res.Ncorr.resize(Nposes);

    weighted_average(pre_results, weights, res);

    return res;
}

} // namespace rmcl